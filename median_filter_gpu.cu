#include "hip/hip_runtime.h"
#include <stdio.h>

extern "C"{
#include "bmp.h"
}

BMPFile* bmp_img; 
unsigned char* host_img_in_data;
unsigned char* host_img_out_data; 
unsigned char* host_test_data;
unsigned char* host_test_data2;  
int blockSize;
int size = 1024 * 1024 * sizeof(char); 

__device__ void _getMask(unsigned char* mask, unsigned char* image_data, int width, int x, int y)
{
	int mask_index = 0; 
	for(int i = x - 1; i < x + 2; i++)
	{
		for(int j = y - 1; j < y + 2; j++)
		{
			mask[mask_index++] = image_data[i * width + j];
		}
	}
}

__device__ void _sort(unsigned char* mask){
	unsigned char temp;
	for(int i = 0; i < sizeof(mask); i++){
		for(int j = 0; j < sizeof(mask); j++){
			if(mask[j] > mask[j+1]){
				temp = mask[j+1]; 
				mask[j+1] = mask[i];
				mask[j] = temp; 
			}
		}
	}
}

void prologue()
{
	host_test_data = (unsigned char*)malloc(size); 
	host_test_data2 = (unsigned char*)malloc(size);
	host_test_data = bmp_img->data;  
	hipMalloc((void**)&host_img_in_data, size); 
	hipMemcpy(host_img_in_data, host_test_data,size, hipMemcpyHostToDevice);	
	
	hipError_t err; 
	hipMalloc((void**)&host_img_out_data, size); 
	hipMemcpy(host_img_out_data, bmp_img->data, size, hipMemcpyHostToDevice);	
	err = hipGetLastError();
	if (hipSuccess != err)
	{
		printf("Prologue failed\n"); 
		exit(1);
	} 	
}

void epilogue()
{
	hipMemcpy(host_test_data2, host_img_in_data, size, hipMemcpyDeviceToHost); 
	hipMemcpy(host_test_data, host_img_out_data, size, hipMemcpyDeviceToHost); 
	hipError_t err; 
	err = hipGetLastError();
	if (hipSuccess != err)
	{
		printf("Epilogue failed\n"); 
		exit(1);
	} 	
	hipFree(host_img_out_data);
	hipFree(host_img_in_data);
}

void _printPixelVals(unsigned char* image_data, int width, int height)
{
	for(int i = 0; i < height; i++){
		for(int j = 0; j < width; j++){
			printf("Img(%d, %d): %02x\n", i, j, image_data[i * height  +j]);
		}
	}
}

__global__ void _applyFilter(unsigned char* host_img_in_data, unsigned char* host_img_out_data, int width, int height)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y; 
	int j = blockIdx.x * blockDim.x + threadIdx.x; 

	unsigned char mask[9]; 
	
	if((i == 0) || (j == 0) || (i == width - 1) || (j == height - 1))
	{
		host_img_out_data[j * width + i] =  0; 
	}
	else 
	{
		_getMask(mask, host_img_in_data, width, i, j);
		_sort(mask);
		host_img_out_data[i * width + j] = mask[4]; 	
	}
}

int main(int argc, char **argv)
{
	FILE *fIn, *fOut; 
	struct hipDeviceProp_t prop;
	hipError_t res; 

	res = hipGetDeviceProperties(&prop, 0); 
	if(hipSuccess != res)
	{
		printf("Loading device properties failed \n");
		exit(1); 
	}

	blockSize = prop.maxThreadsPerBlock; 

	fIn = fopen("indeks.bmp", "rb");
	bmp_img = read_bmp(fIn); 
	fclose(fIn); 
	
	fOut = fopen("out_gpu.bmp", "wb");

	dim3 threadsPerBlock(32, 32);
	dim3 dimGrid((int)ceil((float)bmp_img->header.width_px / (float)32),(int)ceil((float)bmp_img->header.height_px / (float)32)); 
	prologue(); 
	_applyFilter<<<dimGrid, threadsPerBlock>>>(host_img_in_data, host_img_out_data, bmp_img->header.width_px, bmp_img->header.height_px);	
	hipDeviceSynchronize();
	

	epilogue(); 
	_printPixelVals(bmp_img->data, bmp_img->header.width_px, bmp_img->header.height_px); 
	bmp_img->data = host_test_data; 
	write_bmp(bmp_img, fOut); 
	fclose(fOut); 
	free_bmp(bmp_img); 
	return 0; 
}
