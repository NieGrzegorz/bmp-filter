#include "hip/hip_runtime.h"
#include <stdio.h>
extern "C"{
#include "bmp.h"
}
BMPFile* bmp_img; 
unsigned char* host_img_in_data;
unsigned char* host_img_out_data;  
int blockSize;

__device__ void _getMask(unsigned char* mask, unsigned char* image_data, int width, int x, int y)
{
	int mask_index = 0; 
	for(int i = x - 1; i < x + 2; i++)
	{
		for(int j = y - 1; j < y + 2; j++)
		{
			mask[mask_index++] = image_data[i * width + j];
		}
	}
}

__device__ void _sort(unsigned char* mask){
	unsigned char temp;
	for(int i = 0; i < sizeof(mask); i++){
		for(int j = 0; j < sizeof(mask); j++){
			if(mask[j] > mask[j+1]){
				temp = mask[j+1]; 
				mask[j+1] = mask[i];
				mask[j] = temp; 
			}
		}
	}
}

void prologue()
{
	hipMalloc((void**)&host_img_in_data, sizeof(bmp_img->data)); 
	hipMemcpy(host_img_in_data, bmp_img->data, sizeof(bmp_img->data), hipMemcpyHostToDevice);	
	
	hipMalloc((void**)&host_img_out_data, sizeof(bmp_img->data)); 
	hipMemcpy(host_img_out_data, bmp_img->data, sizeof(bmp_img->data), hipMemcpyHostToDevice);	
}

void epilogue()
{
	hipMemcpy(bmp_img->data, host_img_out_data, sizeof(bmp_img->data), hipMemcpyDeviceToHost); 
	hipFree(host_img_out_data);
	hipFree(host_img_in_data);
}

__global__ void _applyFilter(unsigned char* host_img_data, unsigned char* res, int width, int height)
{
	int i = threadIdx.x; 
	int j = threadIdx.y; 

	unsigned char mask[9]; 
	
	if((i != 0) && (j != 0) && (i != width - 1) && (j != height - 1))
	{
		_getMask(mask, host_img_data, width, i, j);
		_sort(mask);
		res[(i-1) * width + (j -1)] = mask[4]; 	
	}
	else 
	{
		res[j * width + i] =  0; 
	}
}

int main(int argc, char **argv)
{
	FILE *fIn, *fOut; 
	struct hipDeviceProp_t prop;
	hipError_t res; 

	res = hipGetDeviceProperties(&prop, 0); 
	if(hipSuccess != res)
	{
		printf("Loading device properties failed \n");
		exit(1); 
	}

	blockSize = prop.maxThreadsPerBlock; 

	fIn = fopen("indeks.bmp", "rb");
	bmp_img = read_bmp(fIn); 
	fclose(fIn); 
	
	fOut = fopen("out_gpu.bmp", "wb");

	dim3 threadsPerBlock(1024, 1024); 
	prologue(); 
	_applyFilter<<<1, threadsPerBlock>>>(host_img_in_data, host_img_out_data, bmp_img->header.width_px, bmp_img->header.height_px);	
	epilogue(); 

	write_bmp(bmp_img, fOut); 
	fclose(fOut); 
	free_bmp(bmp_img); 
	return 0; 
}
