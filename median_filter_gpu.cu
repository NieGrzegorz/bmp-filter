#include "hip/hip_runtime.h"
#include <stdio.h>
extern "C"{
#include "bmp.h"
}
BMPFile* bmp_img; 
unsigned char* host_img_in_data;
unsigned char* host_img_out_data; 
unsigned char* host_test_data; 
int blockSize;

__device__ void _getMask(unsigned char* mask, unsigned char* image_data, int width, int x, int y)
{
	int mask_index = 0; 
	for(int i = x - 1; i < x + 2; i++)
	{
		for(int j = y - 1; j < y + 2; j++)
		{
			mask[mask_index++] = image_data[i * width + j];
		}
	}
}

__device__ void _sort(unsigned char* mask){
	unsigned char temp;
	for(int i = 0; i < sizeof(mask); i++){
		for(int j = 0; j < sizeof(mask); j++){
			if(mask[j] > mask[j+1]){
				temp = mask[j+1]; 
				mask[j+1] = mask[i];
				mask[j] = temp; 
			}
		}
	}
}

void prologue()
{
	host_test_data = (unsigned char*)malloc(sizeof(bmp_img->data)); 
	hipMalloc((void**)&host_img_in_data, sizeof(bmp_img->data)); 
	hipMemcpy(host_img_in_data, bmp_img->data, sizeof(bmp_img->data), hipMemcpyHostToDevice);	
	
	hipMalloc((void**)&host_img_out_data, sizeof(bmp_img->data)); 
	//hipMemcpy(host_img_out_data, bmp_img->data, sizeof(bmp_img->data), hipMemcpyHostToDevice);	
}

void epilogue()
{
	//hipMemcpy(bmp_img->data, host_img_out_data, sizeof(bmp_img->data), hipMemcpyDeviceToHost); 
	hipMemcpy(host_test_data, host_img_out_data, sizeof(bmp_img->data), hipMemcpyDeviceToHost); 
	hipFree(host_img_out_data);
	hipFree(host_img_in_data);
}

void _printPixelVals(unsigned char* image_data, int width, int height)
{
	for(int i = 0; i < height; i++){
		for(int j = 0; j < width; j++){
			printf("Img(%d, %d): %02x\n", i, j, image_data[i * height  +j]);
		}
	}
}

__global__ void _applyFilter(unsigned char* host_img_in_data, unsigned char* host_img_out_data, int width, int height)
{
	int i = blockIdx.x * blockDim.x +threadIdx.x; 
	int j = blockIdx.y * blockDim.y + threadIdx.y; 

	unsigned char mask[9]; 
	
	if((i != 0) && (j != 0) && (i != width - 1) && (j != height - 1))
	{
		_getMask(mask, host_img_in_data, width, i, j);
		_sort(mask);
		host_img_out_data[i * width + j] = mask[4]; 	
	}
	else 
	{
		host_img_out_data[j * width + i] =  0; 
	}
}

int main(int argc, char **argv)
{
	FILE *fIn, *fOut; 
	struct hipDeviceProp_t prop;
	hipError_t res; 

	res = hipGetDeviceProperties(&prop, 0); 
	if(hipSuccess != res)
	{
		printf("Loading device properties failed \n");
		exit(1); 
	}

	blockSize = prop.maxThreadsPerBlock; 

	fIn = fopen("indeks.bmp", "rb");
	bmp_img = read_bmp(fIn); 
	fclose(fIn); 
	
	fOut = fopen("out_gpu.bmp", "wb");

	dim3 threadsPerBlock(1024, 1024); 
	prologue(); 
	_applyFilter<<<1, threadsPerBlock>>>(host_img_in_data, host_img_out_data, bmp_img->header.width_px, bmp_img->header.height_px);	
	hipDeviceSynchronize();
	

	epilogue(); 
	//_printPixelVals(bmp_img->data, bmp_img->header.width_px, bmp_img->header.height_px); 
	bmp_img->data = host_test_data; 
	write_bmp(bmp_img, fOut); 
	fclose(fOut); 
	free_bmp(bmp_img); 
	return 0; 
}
